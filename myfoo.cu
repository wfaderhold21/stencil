
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <time.h>

//#include <common.h>
#define M 10
#define NR_BLOCK 1024

__global__ void compute(const float * a, float * b)
{
    int i = blockIdx.x;
    int j;    

    for (j = 0; j < M; j++) {
        if ((i + j * NR_BLOCK) > 0 && (i + j * NR_BLOCK) < M) {
            b[i + j * NR_BLOCK] = 0.2 * (a[M+((i+j*NR_BLOCK)-1)] + a[M+(i+j*NR_BLOCK)] + a[M+((i+j*NR_BLOCK)+1)] + a[(i+j*NR_BLOCK)] + a[2*M+(i+j*NR_BLOCK)]);
        }
    } 
}

struct params {
    float ** a;
    float ** b;
    float * c;
    float * d;
    float * c_a;
    float * c_b;
    int up, down, j;
    int stop;
    int num_pes;
};
typedef struct params params_t;

void foo(params_t * param)
{
    int j = param->j;
    int up = param->up;
    int down = param->down;
    int num_pes = param->num_pes;
    // above
    if (up != -1 && j == 0) {
         hipMemcpy(param->c_a, 
                   param->c, 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    } else {
        hipMemcpy(param->c_a, 
                   param->a[j - 1], 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    } 
    // middle
    hipMemcpy(&(param->c_a[M]), 
               param->a[j], 
               M * sizeof(float), 
               hipMemcpyHostToDevice);

    // below
    if (down != -1 && j == param->stop - 1) {
        hipMemcpy(&(param->c_a[2 * M]), 
                   param->d, 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    } else {
        hipMemcpy(&(param->c_a[2 * M]), 
                   param->a[j + 1], 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    }
    /*if (!(down != -1 && j == (M / num_pes - 2))) {
        cudaMemcpy(&(param->c_a[2 * M]), 
                   param->a[j + 1], 
                   M * sizeof(float), 
                   cudaMemcpyHostToDevice);
    } else {
        cudaMemcpy(&(param->c_a[2 * M]), 
                   param->d, 
                   M * sizeof(float), 
                   cudaMemcpyHostToDevice);
    }*/
    hipMemcpy(param->c_b, 
               param->b[j], 
               M * sizeof(float), 
               hipMemcpyHostToDevice);

    compute<<<NR_BLOCK, 1>>>(param->c_a, param->c_b);
    hipMemcpy(param->b[j], param->c_b, M * sizeof(float), hipMemcpyDeviceToHost);
}
           /*cudaMemcpy(c_a, a[j - 1], M * sizeof(float), cudaMemcpyHostToDevice);
           cudaMemcpy(&c_a[M], a[j], M * sizeof(float), cudaMemcpyHostToDevice);
           cudaMemcpy(&c_a[2*M], a[j+1], M * sizeof(float), cudaMemcpyHostToDevice);
           
           cudaMemcpy(b[j], c_b, M * sizeof(float), cudaMemcpyDeviceToHost); */
       //}
       
//       printf("[debug] updating a with b\n");
/*       for (j = 1; j < M - 1; j++) {
           for (k = 1; k < M - 1; k++) {
               a[j][k] = b[j][k];
           }
       }*/
/*       #ifdef DEBUG
       printf("[debug output of b]\n");
       for (j = 0; j < M; j++) {
           for (k = 0; k < M; k++) {
               printf("%5.5g ", a[j][k]);
           }
           printf("\n");
       }
       printf("\n\n");
       #endif * DEBUG * */
    //}   
//}
