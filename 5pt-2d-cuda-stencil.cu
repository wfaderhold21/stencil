
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <time.h>

//#define M 8
static const int M = 10;
static const int nr_blocks = 1024;

static inline struct timespec mydifftime(struct timespec start, struct timespec end)
{
    struct timespec temp;
    if((end.tv_nsec-start.tv_nsec) < 0) {
        temp.tv_sec = end.tv_sec - start.tv_sec - 1;
        temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
    } else {
        temp.tv_sec = end.tv_sec - start.tv_sec;
        temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    }
    return temp;
}

__global__ void compute(const float * a, float * b)
{
    int i = blockIdx.x;
    int j;    

    for (j = 0; j < M; j++) {
        if ((i + j * nr_blocks) > 0 && (i + j * nr_blocks) < M) {
            b[i + j * nr_blocks] = 0.2 * (a[M+((i+j*nr_blocks)-1)] + a[M+(i+j*nr_blocks)] + a[M+((i+j*nr_blocks)+1)] + a[(i+j*nr_blocks)] + a[2*M+(i+j*nr_blocks)]);
        }
    } 
}

int main(int argc, char ** argv) {
    float ** a, ** b, * c;
    float * c_a, * c_b;
    int i = 0;
    int j = 0, k = 0;
    struct timespec time1;
    struct timespec time2;
    struct timespec result;

    a = (float **) malloc(sizeof(float *) * M);
    b = (float **) malloc(sizeof(float *) * M);
    hipMalloc((void **)&c_a, sizeof(float) * M * M);
    hipMalloc((void **)&c_b, sizeof(float) * M * M);
    for (j = 0; j < M; j++) {
        a[j] = (float *) malloc(sizeof(float) * M);
        b[j] = (float *) malloc(sizeof(float) * M);
        
        memset(a[j], 0, sizeof(float) * M);
        memset(b[j], 0, sizeof(float) * M);
    }

    for (j = 0; j < M; j++) {
        a[j][0] = 1;
    }
    for (j = 0; j < M; j++) {
        a[j][M-1] = 1;
    } 
    
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

#ifdef DEBUG
    printf("[debug]\n");
    for (j = 0; j < M; j++) {
        for (k = 0; k < M; k++) {
            printf("%g ", a[j][k]);
        }
        printf("\n");
    }
    printf("\n\n");
#endif /* DEBUG */

    for (i = 0;i < 10;i++) {
#ifdef DEBUG
        printf("Iter: %d\n", i);
        fflush(stdout);
#endif
        for (j = 1; j < M - 1; j++) {
            hipMemcpy(c_a, a[j - 1], M * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(&c_a[M], a[j], M * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(&c_a[2*M], a[j+1], M * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(c_b, b[j], M * sizeof(float), hipMemcpyHostToDevice);
            
            compute<<<nr_blocks, 1>>>(c_a, c_b);
            hipMemcpy(b[j], c_b, M * sizeof(float), hipMemcpyDeviceToHost);
        }
        
        //printf("[debug] updating a with b\n");
        for (j = 1; j < M - 1; j++) {
            for (k = 1; k < M - 1; k++) {
                a[j][k] = b[j][k];
            }
        }
        #ifdef DEBUG
        printf("[debug output of b]\n");
        for (j = 0; j < M; j++) {
            for (k = 0; k < M; k++) {
                printf("%5.5g ", a[j][k]);
            }
            printf("\n");
        }
        printf("\n\n");
        #endif /* DEBUG */ 
    } 
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);                                
    result = mydifftime(time1, time2);
    printf("timing: %lu.%.0f sec\n", result.tv_sec, (float)(result.tv_nsec / 1000000.0));

    free(a);
    free(b);
    hipFree(c_a);
    hipFree(c_b);

    return 0;
}
